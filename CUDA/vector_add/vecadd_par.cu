
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 512000

__global__ void add(int *a, int *b, int *c) {
  // blockIdx is a predefined block /grid variable part of CUDA runtime.

  // blockIdx.x is another runtime different constant. 
  // It points to the current index of the tensor.
  c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void random_ints(int *r, int n) {
  int i;
  for (i = 0; i < n; i++) {      
    r[i] = rand()%5000;
  }
}

int main(void) {

  int *a, *b, *c;
  // host copies of a, b, c
  int *d_a, *d_b, *d_c;
  // device copies of a, b, c
  int size = N * sizeof(int);

  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // Setup input values
  a = (int *)malloc(size); 
  random_ints(a, N);
  b = (int *)malloc(size); 
  random_ints(b, N);
  c = (int *)malloc(size);

  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  // Launch add() kernel on GPU with N Blocks on device. With block size of 1.
  add<<<N,1>>>(d_a, d_b, d_c);

  // Copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  for (int i = 0; i < N; i++) {
    printf("(a, b, c) = (%d, %d, %d)\n", a[i], b[i], c[i]);
  }

  // Cleanup
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  free(a); free(b); free(c);


  return 0;
}
